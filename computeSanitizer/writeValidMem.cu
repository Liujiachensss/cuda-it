
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#define N 1023
__global__ void scaleArray(float* array, float value) {
  int threadGlobalID    = threadIdx.x + blockIdx.x * blockDim.x;
  array[threadGlobalID] = array[threadGlobalID]*value;
  return;
}

int main() {
  float* array;
  hipMallocManaged(&array, N*sizeof(float)); 
  for(int i=0; i<N; i++) array[i] = 1.0f;    
  printf("Before: Array 0, 1 .. N-1: %f %f %f\n", array[0], array[1], array[N-1]);
  scaleArray<<<4, 256>>>(array, 3.0);
  hipDeviceSynchronize();
  printf("After : Array 0, 1 .. N-1: %f %f %f\n", array[0], array[1], array[N-1]);
  assert(array[N/2] == 3.0); 
  exit(0);
}